#include "hip/hip_runtime.h"
#include <stdint.h>

#include "sha1.h"

__constant__ uint32_t c_block[16];
__constant__ hash_digest_t c_ctx;
__constant__ uint32_t c_difficulty[5];

__device__ __forceinline__ uint32_t f1(uint32_t b, uint32_t c, uint32_t d){
    return (b & c) | ((~b) & d);
}

__device__ __forceinline__ hash_digest_t computeSHA1Block(uint32_t* in, uint32_t id, uint32_t idx,
                                                     hash_digest_t* h)
{
    uint32_t a = h->h0;
    uint32_t b = h->h1;
    uint32_t c = h->h2;
    uint32_t d = h->h3;
    uint32_t e = h->h4;
    uint32_t f;
    uint32_t k;
    uint32_t temp;
    uint32_t w[16];
    int i;

#pragma unroll 11
    for (i = 0; i < 11; ++i) {
        w[i] = in[i];
    }

    w[11] = idx;
    w[12] = id;

#pragma unroll 3
    for (i = 13; i < 16; ++i) {
        w[i] = in[i];
    }

    k = 0x5A827999;
    //0 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //1 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //2 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //3 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //4 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //5 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //6 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //7 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //8 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //9 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //10 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //11 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //12 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //13 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //14 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //15 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //16 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //17 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //18 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //19 of 0-20
    temp = ((a << 5) | (a >> 27)) + f1(b, c, d) + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    k = 0x6ED9EBA1;
    //20 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //21 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //22 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //23 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //24 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //25 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //26 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //27 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //28 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //29 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //30 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //31 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //32 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //33 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //34 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //35 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //36 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //37 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //38 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //39 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    k = 0x8F1BBCDC;
    //40 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //41 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //42 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //43 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //44 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //45 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //46 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //47 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //48 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //49 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //50 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //51 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //52 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //53 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //54 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //55 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //56 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //57 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //58 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //59 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    k = 0xCA62C1D6;

    //60 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //61 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //62 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //63 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;


    //64 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //65 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //66 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //67 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //68 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //69 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //70 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //71 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //72 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //73 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //74 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //75 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //76 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //77 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //78 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //79 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    return (hash_digest_t) { 
        h->h0 + a,
        h->h1 + b,
        h->h2 + c,
        h->h3 + d,
        h->h4 + e
    };
}

__global__ void
__launch_bounds__(THREADS_PER_BLOCK) // Add BLOCKS_PER_SM arg to limit register usage
shaforce(volatile uint32_t* result,
                         const __restrict__ uint32_t idx)
{
    uint8_t i;
    hash_digest_t res;
    uint32_t global_id = blockIdx.x * blockDim.x + threadIdx.x;

    for(i = 0; i < 16; ++i){
        global_id |= (i << 24);

        res = computeSHA1Block(c_block, global_id, idx, &c_ctx);
        uint8_t found = 0;
        if((res.h0 < c_difficulty[0]) ||
           (res.h0 == 0 && res.h1 < c_difficulty[1])) {
            printf("DIFFICULTY: %08x,%08x < %08x,%08x\n", res.h0, res.h1, c_difficulty[0], c_difficulty[1]);
            // Add one so zero can signal not-found
            atomicMax((uint32_t*)result, global_id+1);
            break;
        }/* else if(*result){
            break;
            }*/

        global_id &= 0x00ffffff;
    }
}


extern "C" void force_kernel(unsigned int *d_result,
                             const uint32_t idx){
    shaforce<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_result, idx);
}

extern "C" hipError_t copy_constants(uint32_t *h_block,
                                      uint8_t *h_difficulty,
                                      hash_digest_t *h_ctx){
    return (hipError_t)(
        hipMemcpyToSymbol(HIP_SYMBOL(c_block), h_block, sizeof(uint32_t) * 16) |
        hipMemcpyToSymbol(HIP_SYMBOL(c_difficulty), h_difficulty, sizeof(uint32_t) * 5) |
        hipMemcpyToSymbol(HIP_SYMBOL(c_ctx), h_ctx, sizeof(hash_digest_t)));
}
